/* All the source */

#include <hip/hip_runtime.h>
#include <iostream> // Cin/ Cout
#include <fstream>  // For Read/Write File
#include <string>
#include <sstream>  // using sstream
#include <vector>   // For using Vector
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
using namespace std;
#define TILE_DIM 32    // For 2D Thread Block Allocation 
#define Block_Size 128 // For 1D Thread Block Allocation 
enum run_mode {GPU_PARA=0, CPU_GOLDEN, READ_THRU};

//int parse_cmd_line(int argc, char* argv[], string & G_File_Name, string & K_File_Name, int & Individual_Num, int & MarkerBlock_Size);

int parse_cmd_line(int argc, char* argv[], string & G_File_Name, string & K_File_Name, int & Individual_Num, int & MarkerBlock_Size, char & Run_Mode)
{ 
	if((argc!=2)&&(argc <11))
	{
		cerr <<"Please specific the correct parameters, or use parameter -u for usermanuals!" <<endl;
		return 1;
	}
	else
	{
	   for (int i=1; i<argc;i++)
       {
	       string arg=argv[i];
	       if((arg=="-g")||(arg=="-G"))
	       {
             if(i+1<argc)
		     {
		       G_File_Name =argv[i+1];
		     }
		     else
		     {
			   cerr<<"Parse cmd_line fail, Need to clearly specific the full file name of genotype matrix !" <<endl;
			   return 1; // Parse cmd_line fail
		      }
		    }	
		    
		   if((arg=="-k")||(arg=="-K"))
	       {
             if(i+1<argc)
		     {
		       K_File_Name =argv[i+1];
		     }
		     else
		     {
			   cerr<<"Parse cmd_line fail, Need to clearly specific the full file name of the output kinship matrix !" <<endl;
			   return 1; // Parse cmd_line fail
		      }
		    }        

            if((arg=="-i")||(arg=="-I"))
	        {
              if(i+1<argc)
		      {
		           Individual_Num =atoi(argv[i+1]);
		      }
		      else
		      {
			   cerr<<"Parse cmd_line fail, Need to clearly specific the Individual Number!" <<endl;
			   return 1; // Parse cmd_line fail
		      }
	        }
	        
	        if((arg=="-b")||(arg=="-B"))
	        {
              if(i+1<argc)
		      {
		           MarkerBlock_Size =atoi(argv[i+1]);
		      }
		      else
		      {
			   cerr<<"Parse cmd_line fail, Need to clearly specific the Marker Block Size!" <<endl;
			   return 1; // Parse cmd_line fail
		      }
	        }
	        
	        if((arg=="-m")||(arg=="-M"))
	        {
              if(i+1<argc)
		      {
		           Run_Mode =atoi(argv[i+1]);
		      }
		      else
		      {
			   cerr<<"Parse cmd_line fail, Need to clearly specific the Run Mode(0: GPU Paral; 1: Golden CPU; 2: Read thorugh) !" <<endl;
			   return 1; // Parse cmd_line fail
		      }
	        } 

		   if((arg=="-h")||(arg=="-H"))
	       {
			     cout<<"Welcome to use this program to do Kinship Matrix Calculation" <<endl;
				 cout << "The usage of input parameter arguments are listed as followings:" <<endl;
				 cout << "-h or -H: Output this Help usage message" <<endl;
				 cout << "-g or -G: The full name of Genotype file"<<endl;
				 cout << "-k or -K: The full name of Kinship Matrix file"<<endl;
				 cout << "-i or -I: The Individual number" <<endl;
				 cout << "-b or -B: The Block size for dividing the Genotype markers" <<endl;
				 cout << "-m or -M: Run_Mode(0, 1, 2 for the GPU/CPU/Read Through)" <<endl;
			     return 1; // Parse cmd_line fail
		    }
	    }
	}
    return 0;
}

__global__ void Genotype_Matrix_Transpose(float *GMatrix_O, float *GMatrix_I, int Matrix_Width, int Matrix_Height) //ok
{
   __shared__ float block[TILE_DIM][TILE_DIM+1];
   int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
   int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
   int Index_In = xIndex + (yIndex)*Matrix_Width;
    
   if( (xIndex < Matrix_Width) && (yIndex < Matrix_Height))
      block[threadIdx.y][threadIdx.x] = GMatrix_I[Index_In];
   else    
      block[threadIdx.y][threadIdx.x] = 0.0; //pad zero for the unarbitrary matrix 
          
   __syncthreads();
    
   xIndex = blockIdx.y * TILE_DIM + threadIdx.y;
   yIndex = blockIdx.x * TILE_DIM + threadIdx.x;
   int Index_Out = xIndex + (yIndex)*Matrix_Height;
   if( (xIndex < Matrix_Height)&&(yIndex < Matrix_Width))
      GMatrix_O[Index_Out] = block[threadIdx.y][threadIdx.x];     
      
   __syncthreads();  
     
}

__global__ void Kinship_Matrix_Cal(float *Genotype_Matrix_Transpose_In, float *Genotype_Matrix_In, float *Kinship_Matrix_Out,int Individual_Num,  int Marker_Num) 
{
   /* Use shared memory to calculate t(GM)*GM, 
    * GM is original genotype matrix, which is a mxn matrix and m is marker number; 
    * t(GM) means the transpose of the original genotype matrix, which is a nXm matrix  */
      
   __shared__ float t_s[TILE_DIM][TILE_DIM];   // shared memory for transposed matrix
   __shared__ float m_s[TILE_DIM][TILE_DIM];   // shared memory for matrix 
   
   int r_t     = TILE_DIM* blockIdx.y+ threadIdx.y; // row index for transposed matrix 
   int c_t     = threadIdx.x;                       // column index for transposed matrix
   int r_m     = threadIdx.y;                       // row index of matrix
   int c_m     = blockIdx.x* TILE_DIM+  threadIdx.x;// column index for matrix                  
   
   float entry_sub =0.0;  
   for(int i=0; i< (Marker_Num +TILE_DIM-1)/TILE_DIM; i++)
   {
	   float value_t, value_m;
	   int index_t = r_t*Marker_Num +c_t;         //linear index of transposed matrix
	   int index_m = r_m* Individual_Num+ c_m;    //linear index of genotype matrix
	   
	   if ((r_t < Individual_Num )&&(c_t < Marker_Num))	   
	       value_t = Genotype_Matrix_Transpose_In[index_t];	   
	   else	   
		   value_t =0.0;  // pad with 0
	   
	   if ((r_m< Marker_Num)&&(c_m < Individual_Num))
	       value_m = Genotype_Matrix_In[index_m];
	   else
	       value_m =0.0;  // pad with 0
	   
	   c_t += TILE_DIM;
	   r_m += TILE_DIM;    
	   t_s[threadIdx.y][threadIdx.x]  = value_t; 
	   m_s[threadIdx.y][threadIdx.x]  = value_m;	   
	   __syncthreads();
	   
	   for (int k=0; k< TILE_DIM; k++)
	     entry_sub += t_s[threadIdx.y][k]* m_s[k][threadIdx.x];
	     
	   __syncthreads();
	   	  
   }
   
   int r_k     = TILE_DIM* blockIdx.y+ threadIdx.y; // row index for kinship matrix;
   int c_k     = TILE_DIM* blockIdx.x+ threadIdx.x; // column index for kinship matrix;
   int index_k = r_k* Individual_Num+ c_k;          //linear index of kinship matrix
   
   if((r_k<Individual_Num) &&(c_k<Individual_Num))
	   Kinship_Matrix_Out[index_k] =entry_sub;
	   
   __syncthreads();
  
}

__global__ void Kinship_Matrix_Add(float *Kinship_Matrix_Sub, float *Kinship_Matrix, long Matrix_Size) 
{  
   int index  = blockIdx.x * blockDim.x + threadIdx.x;
   long stride = blockDim.x * gridDim.x; 
   for (long i = index; i < Matrix_Size; i += stride)
   Kinship_Matrix[i] = Kinship_Matrix[i] +  Kinship_Matrix_Sub[i];
   
}

__global__ void Kinship_Matrix_Normalize(float *Kinship_Matrix, long Matrix_Size, float Normalize_Rate) 
{
   int index  = blockIdx.x * blockDim.x + threadIdx.x;
   long stride = blockDim.x * gridDim.x; 
   for (long i = index; i < Matrix_Size; i += stride)
      Kinship_Matrix[i] = Kinship_Matrix[i]/Normalize_Rate ;
}

__global__ void Matrix_AllValue_Set(float *Matrix, long Matrix_Size, float Value_Set) 
{
   int index  = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x; 
   
   for (long i = index; i < Matrix_Size; i += stride)
       Matrix[i] = Value_Set;
}

/*The golden code implementation of kinship matrix calculoation, which can be used for the base evaluation of GPU paralleling*/
void Golden_Kinship_Matrix_Cal(float *Kinship_Matrix_Out, float *GMatrix_I,  int Individual_Num,  int Marker_Num)
{
	for (int i =0; i< Individual_Num; i++)
	{		
	   for (int j=0; j< Individual_Num; j++)
	   {  
		   float sum =0.0;
		   for (int m=0; m< Marker_Num; m++)
		   { 
			  	float temp1 = *(GMatrix_I+m*Individual_Num +i);
			  	float temp2 = *(GMatrix_I+m*Individual_Num +j); 
			  	sum += temp1 * temp2;			   
		   }
		   *(Kinship_Matrix_Out+i* Individual_Num+j) = sum; 		   
	   }
    }
}

void Golden_Kinship_Matrix_Add (float *Kinship_Matrix_Sub, float *Kinship_Matrix, long Matrix_Size)
{ 
	 for (long i = 0; i < Matrix_Size; i ++)
     Kinship_Matrix[i] = Kinship_Matrix[i] +  Kinship_Matrix_Sub[i];
}


void Golden_Kinship_Matrix_Normalize(float *Kinship_Matrix, long Matrix_Size, float Normalize_Rate) 
{  
   for (long i = 0; i < Matrix_Size; i ++)
      Kinship_Matrix[i] = Kinship_Matrix[i]/Normalize_Rate ;
}

void Golden_Matrix_AllValue_Set(float *Matrix, long Matrix_Size, float Value_Set) 
{   
   for (long i = 0; i < Matrix_Size; i ++)
       Matrix[i] = Value_Set;
}

int main(int argc, char* argv[])
{
   clock_t c_begin, c_end;
   c_begin=clock();
 
   // begin to calculate the time
   string G_File_Name, K_File_Name;   
   char Run_Mode;
   int Individual_Num, MarkerBlock_Size;
   string OS_Path_Sep="//";
   long read_line_count=0;

   // Parse the command line for the inputting
   if(1==parse_cmd_line(argc, argv, G_File_Name, K_File_Name, Individual_Num, MarkerBlock_Size, Run_Mode))
        return 1;
  
   //Open and Read the Geneotype Matrix Marker_Block one by one.
	ifstream G_File(G_File_Name.c_str(), ios::in);

    if(!G_File.is_open())
	{
		cerr<< G_File_Name <<" Can't be accessed!"<<endl;
		return 1;
	}
		
	if(G_File)
    {
       float *G_Matrix, *G_Matrix_Tran, *KinshipMatrix_Sub, *KinshipMatrix;
       string sLine;
       const long g_matrix_size =  Individual_Num*MarkerBlock_Size;
       const long k_matrix_size =  Individual_Num*Individual_Num;
       
       switch (Run_Mode)
       {
         case GPU_PARA :
            hipMallocManaged(&G_Matrix, sizeof(float)*g_matrix_size);
            hipMallocManaged(&G_Matrix_Tran, sizeof(float)*g_matrix_size);
            hipMallocManaged(&KinshipMatrix_Sub, sizeof(float)*k_matrix_size);
            hipMallocManaged(&KinshipMatrix, sizeof(float)*k_matrix_size);
            break;
         default :  
		    G_Matrix          = new float[g_matrix_size]; 
		    KinshipMatrix_Sub = new float[k_matrix_size];
		    KinshipMatrix     = new float[k_matrix_size];
		    break;
	   }
       	   
	   dim3 thread2ds(TILE_DIM,TILE_DIM);
       dim3 grid2ds((Individual_Num+TILE_DIM-1)/TILE_DIM, (MarkerBlock_Size+TILE_DIM-1)/TILE_DIM);
       dim3 grid2ds1((Individual_Num+TILE_DIM-1)/TILE_DIM, (Individual_Num+TILE_DIM-1)/TILE_DIM);
       dim3 thread1ds(Block_Size);
	   dim3 grid1ds((Individual_Num*Individual_Num)/Block_Size);
	   
	   int Line_Count=0;
	   while(getline(G_File, sLine))
	   {
		  if(sLine.empty()) ; // Ignore empty lines
		  else
		  {
            stringstream ss(sLine);
			vector <string> s_v;
			string item;
			char delim1 =','; 
			char delim2 ='\t';
		
			while(getline(ss, item, delim1))
			{
				s_v.push_back(item);
			}

			int Col_Num=s_v.size(); 
			if (Col_Num < Individual_Num)
			{
				s_v.clear(); 
				while(getline(ss, item, delim2))// try delim2;
			    {
				   s_v.push_back(item);
			    }
			    Col_Num=s_v.size(); 				
			}
			
			if(Col_Num != Individual_Num) 
			{
				cerr<< G_File_Name <<"File Format is not right, Error at Line=" << Line_Count<<endl;
		        return 1;
			}
			
			for (int i=0; i<Col_Num;i++)
			{
			   float value= atof(s_v.at(i).c_str());
			     G_Matrix[Line_Count*Individual_Num+i] = value;
			}
          }
		  Line_Count++;
		  
		  if(Line_Count< MarkerBlock_Size) 
		     continue;
		  else
		  { 			 
			 switch(Run_Mode)
			 {
			    case GPU_PARA :
			       // begin to call the GPU program to calculate the partial kinship matrix			  
                   Genotype_Matrix_Transpose<<<grid2ds, thread2ds >>>(G_Matrix_Tran, G_Matrix, Individual_Num, MarkerBlock_Size);
                   hipDeviceSynchronize();// Wait for GPU to finish before accessing on host
             
                   Kinship_Matrix_Cal<<<grid2ds1, thread2ds >>>(G_Matrix_Tran, G_Matrix, KinshipMatrix_Sub, Individual_Num, MarkerBlock_Size);  
                   hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host
			 
			       // add the block kinship matrix into the all kinship matrix
			       Kinship_Matrix_Add <<<grid1ds, thread1ds>>>(KinshipMatrix_Sub,  KinshipMatrix, k_matrix_size); 		  
		           hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host
					 
			       Matrix_AllValue_Set <<<grid1ds, thread1ds>>> (G_Matrix, g_matrix_size, 0.0);
			       hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host		
			       break;	
			    case CPU_GOLDEN:
			       Golden_Kinship_Matrix_Cal(KinshipMatrix_Sub, G_Matrix,  Individual_Num,  MarkerBlock_Size);
			       Golden_Kinship_Matrix_Add(KinshipMatrix_Sub, KinshipMatrix, k_matrix_size);
			       Golden_Matrix_AllValue_Set(G_Matrix, g_matrix_size, 0.0); 
			       break;
			    default :
			       read_line_count +=Line_Count;
			       break;  			    
     
			}
		    Line_Count =0;  // Reset the Line_Count to 0
		 }   
       }
       
       if(Line_Count>0) //  At the rear part of the input large scale genotype file, we have read some residual lines 
       { 
		   
		   switch(Run_Mode)
		   {
			 case GPU_PARA :  
		       
                Genotype_Matrix_Transpose<<<grid2ds, thread2ds >>>(G_Matrix_Tran, G_Matrix, Individual_Num, MarkerBlock_Size);
                hipDeviceSynchronize();// Wait for GPU to finish before accessing on host
                        
                Kinship_Matrix_Cal<<<grid2ds1, thread2ds >>>(G_Matrix_Tran, G_Matrix, KinshipMatrix_Sub, Individual_Num,  MarkerBlock_Size);  
                hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host
			 
		        // add the block kinship matrix into the all kinship matrix
		        Kinship_Matrix_Add <<<grid1ds, thread1ds>>>(KinshipMatrix_Sub,  KinshipMatrix, k_matrix_size); 
		        hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host
		        break;
		        
		     case CPU_GOLDEN:
			    Golden_Kinship_Matrix_Cal(KinshipMatrix_Sub, G_Matrix,  Individual_Num,  MarkerBlock_Size);
			    Golden_Kinship_Matrix_Add(KinshipMatrix_Sub, KinshipMatrix, k_matrix_size);
			    Golden_Matrix_AllValue_Set(G_Matrix, g_matrix_size, 0.0);   
		        break;
		        
			  default:
			    read_line_count +=Line_Count;
			    break;   
		   }		   
		   Line_Count =0;  // Reset the Line_Count to 0 
	   }
	   
	   // Begin to calculate the Normalized ratio 
	   float Normalize_Rate =0.0;
	   for (int i_index =0; i_index < Individual_Num; i_index++)
	    Normalize_Rate += KinshipMatrix[i_index*Individual_Num+i_index];
	  
	   Normalize_Rate =Normalize_Rate/Individual_Num;
	   
	   switch(Run_Mode)
	   {
	      case GPU_PARA : 	      
	        Kinship_Matrix_Normalize <<<grid1ds, thread1ds>>>(KinshipMatrix, k_matrix_size, Normalize_Rate) ;
	        hipDeviceSynchronize(); // Wait for GPU to finish before accessing on host
	        break;
	      case CPU_GOLDEN:
	        Golden_Kinship_Matrix_Normalize(KinshipMatrix, k_matrix_size, Normalize_Rate);
	        break;
	      default :
	        break;
	   }
	   
	   //Output the kinship matrix
	   ofstream Kinship_File; // used to output the calculated kinship matrix 
	   Kinship_File.open(K_File_Name.c_str());
	   if(Kinship_File.is_open())
	   {
			if(Run_Mode <READ_THRU)
			{
			   char delim ='\t';			
			   for (int i_individual=0; i_individual < Individual_Num; i_individual++)
			   {
			     Kinship_File<<KinshipMatrix[i_individual*Individual_Num]; 
			     for (int j_individual=1; j_individual < Individual_Num; j_individual++)
			     {
			        Kinship_File<< delim<< KinshipMatrix[i_individual*Individual_Num + j_individual];
			     }
			     Kinship_File<<endl;  // end line			   
			   }
		    }
		    else
		    {
				Kinship_File << "Read through, Read_Line_Count is:" << read_line_count << endl;
			}
		}
		else
		{
			cerr <<"Error open the result file for the output Kinship Matrix!"<<endl;
		}

		Kinship_File.close();
	   
	    // Free the allocated matrix 	
	    switch (Run_Mode)
	    {  
		   case GPU_PARA : 
              hipFree(G_Matrix);
              hipFree(G_Matrix_Tran);
              hipFree(KinshipMatrix_Sub);
              hipFree(KinshipMatrix);  
              break;
           default:   	    
		      delete []G_Matrix; 
		      delete []KinshipMatrix_Sub;
		      delete []KinshipMatrix;
		      break;
		}     
	} 

   c_end =clock();
   double elapse_time = double (c_end-c_begin)/CLOCKS_PER_SEC; 
   cout << "Hello, the elapse time is " <<  elapse_time << " seconds" << endl;
  /*
  
   const int size_x =2048, size_y = 2048;  //2048, 2048, 1<<15
   dim3 grid(size_x/TILE_DIM, size_y/BLOCK_ROWS);
   dim3 threads(TILE_DIM,BLOCK_ROWS);
 //  dim3 threads(TILE_DIM,TILE_DIM);
     
   const long mem_size = sizeof(float) * size_x*size_y;
   float *d_idata, *d_odata;
	
   // Allocate Unified Memory – accessible from CPU or GPU
   
   cudaMallocManaged(&d_idata, mem_size);
   cudaMallocManaged(&d_odata, mem_size);
  
   // initalize host data
   for(long i = 0; i < (size_x*size_y); ++i)
   d_idata[i] = (float) i;
    
/*  
 //  clock_t c_begin, c_end;
 //  c_begin=clock();
 //  matrix_transpose_gold(d_odata, d_idata, size_x, size_y, NUM_REPS);
 //  c_end =clock();
 //  double cpu_elapse_time = double (c_end-c_begin)/CLOCKS_PER_SEC;
 // 
    
    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    cudaEventRecord(start, 0);
   // cudaSetDevice(1); 
   
   
    matrix_transpose1<<<grid, threads >>>(d_odata, d_idata, size_x, size_y, NUM_REPS);
  
   cudaEventRecord(stop, 0);
   cudaEventSynchronize(start);
   cudaEventSynchronize(stop);
   
   float Elapse_Time ;
   cudaEventElapsedTime(&Elapse_Time,start, stop);

  // Wait for GPU to finish before accessing on host
  cudaDeviceSynchronize();
  
  
  float maxError = 0.0f;
  for (int i = 0; i < size_y; i++)
  { 
	   for (int j = 0; j < size_x; j++)
	   {
		   maxError = fmax(maxError, fabs(d_idata[i*size_x+j]-d_odata[j*size_y+i]));
		   if(maxError>0.1)
		     cout << "i=" <<i << "j=" <<j << "d_idata(i,j)="  << d_idata[i*size_x+j] << "d_odata(j,i)=" << d_odata[j*size_y+i]<< endl;
	   }
  }
   
   
  std::cout << "Max error: " << maxError << std::endl;
  
  //std::cout <<"d_idata[0,0]="<< d_idata[0] <<"d_idata[0,1]=" <<d_idata[1] << "d_idata[2048]="<< d_idata[2048] <<"d_idata[2049]=" <<d_idata[2049]<< std::endl; 
  //std::cout <<"d_odata[0,0]="<< d_odata[0] <<"d_odata[0,1]=" <<d_odata[1] << "d_odata[2048]="<< d_odata[2048] <<"d_odata[2049]=" <<d_odata[2049]<< std::endl; 

  
  cudaFree(d_idata);
  cudaFree(d_odata);

 cout << "Hello, the elaspe Time in GPU is " <<  Elapse_Time << endl;
 //  cout <<"Hello, the CLOCK NUMBER CPU is" << int(c_end-c_begin) <<endl;
 // cout <<"Hello, the elaspe time in CPU is" << cpu_elapse_time <<endl;
  return 0;
  
  */
}

void test_matrix_transpose() 
{
   /* The following are GPU interface for testing the GPU version for Matrix Transpose at arbitrary dimension*/ 
  
   int Matrix_Width  =10000;
   int Matrix_Height =50000;  
   dim3 threads(TILE_DIM,TILE_DIM);
   dim3 grids((Matrix_Width+TILE_DIM-1)/TILE_DIM, (Matrix_Height+TILE_DIM-1)/TILE_DIM);
   
   const long mem_size = sizeof(float)*Matrix_Width*Matrix_Height;
   float *GMatrix_O, *GMatrix_I;
   hipMallocManaged(&GMatrix_O, mem_size);
   hipMallocManaged(&GMatrix_I, mem_size);
  
   // initalize host data
   for(long i = 0; i < (Matrix_Width*Matrix_Height); ++i)
   GMatrix_I[i] = (float) i;
     
   Genotype_Matrix_Transpose<<<grids, threads >>>(GMatrix_O, GMatrix_I, Matrix_Width, Matrix_Height);
  
   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();
  
   
   float maxError = 0.0f;
   for (int i = 0; i < Matrix_Height; i++)
   { 
	   for (int j = 0; j < Matrix_Width; j++)
	   {
		   maxError = fmax(maxError, fabs(GMatrix_I[i*Matrix_Width+j]-GMatrix_O[j*Matrix_Height+i]));
		   if(maxError>0.1)
		     cout << "i=" <<i << "j=" <<j << "GMatrix_I(i,j)="  << GMatrix_I[i*Matrix_Width+j] << "GMatrix_O(j,i)=" << GMatrix_O[j*Matrix_Height+i]<< endl;
	   }
   }
     
  std::cout << "Max error: " << maxError << std::endl;
   
  hipFree(GMatrix_O);
  hipFree(GMatrix_I);
  return ; 	
}

void test_kinship()
{
	  /* The following are GPU interface for testing the GPU version for kinship matrix calculating by multiplying the transpose of genotype Matrix and genotype matrix at arbitrary dimension*/ 
   
   int Marker_Num =1000;
   int Individual_Num =2500;
    
   dim3 threads(TILE_DIM,TILE_DIM);
   dim3 grids((Individual_Num+TILE_DIM-1)/TILE_DIM, (Marker_Num+TILE_DIM-1)/TILE_DIM);
   
   const long genotype_matrix_size = sizeof(float)*Marker_Num*Individual_Num;
   const long kinship_matrix_size = sizeof(float)* Individual_Num*Individual_Num;
   float *Genotype_Matrix_T,  *Genotype_Matrix,  *Kinship_Matrix;
   hipMallocManaged(&Genotype_Matrix_T, genotype_matrix_size);
   hipMallocManaged(&Genotype_Matrix, genotype_matrix_size);
   hipMallocManaged(&Kinship_Matrix, kinship_matrix_size);
    
   // initalize host data
   for(long i = 0; i < (Marker_Num*Individual_Num); ++i)
   Genotype_Matrix[i] = (float) 1.0*i/(Marker_Num);
     
   Genotype_Matrix_Transpose<<<grids, threads >>>(Genotype_Matrix_T, Genotype_Matrix, Individual_Num, Marker_Num);
  
   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();
   
      
   float maxError = 0.0f;
   for (int i = 0; i < Marker_Num; i++)
   { 
	   for (int j = 0; j < Individual_Num; j++)
	   {
		   maxError = fmax(maxError, fabs(Genotype_Matrix[i*Individual_Num+j]-Genotype_Matrix_T[j*Marker_Num+i]));
		   if(maxError>0.1)
		     cout << "i=" <<i << "j=" <<j << "Genotype_Matrix(i,j)="  << Genotype_Matrix[i*Individual_Num+j] << "Genotype_Matrix_T(j,i)=" << Genotype_Matrix_T[j*Marker_Num+i]<< endl;
	   }
   }
     
  std::cout << "Matrix Transpose Max error: " << maxError << std::endl;
  
  dim3 grids1((Individual_Num+TILE_DIM-1)/TILE_DIM, (Individual_Num+TILE_DIM-1)/TILE_DIM);
  Kinship_Matrix_Cal<<<grids1, threads >>>(Genotype_Matrix_T, Genotype_Matrix, Kinship_Matrix, Marker_Num, Individual_Num);
  
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  std::cout <<"Kinship_Matrix[0,0]="<< Kinship_Matrix[0] <<"Kinship_Matrix[0,1]=" <<Kinship_Matrix[1] << "Kinship_Matrix[0,1023]="<< Kinship_Matrix[1023] <<"Kinship_Matrix[1,1]=" <<Kinship_Matrix[1024]<< std::endl; 
 
  hipFree(Genotype_Matrix_T);
  hipFree(Genotype_Matrix);
  hipFree(Kinship_Matrix); 
  return;
}

void test_mis()
{
/*	  std::cout << "g_matrix(0,0)=" <<  G_Matrix[0]<< std::endl;
			 cudaMemset(G_Matrix, 0, g_mem_size*sizeof(float));
			 cudaMemset(KinshipMatrix_Sub, 0, k_mem_size*sizeof(float));
	  std::cout << "Line_Count= " << Line_Count << std::endl;
			 std::cout << "g_matrix(0,0)=" <<  G_Matrix[0]<< std::endl;
			 std::cout << "kmatrix_sub(0,0)=" <<  KinshipMatrix_Sub[0]<< std::endl;
			 std::cout << "kmatrix(0,0)=" <<  KinshipMatrix[0]<< std::endl;
			 std::cout << "Line_Count= " << Line_Count << std::endl;	  
		   std::cout << "kmatrix_sub( 0,0)=" <<  KinshipMatrix_Sub[0]<< std::endl;
		   std::cout << "kmatrix(0,0)=" <<  KinshipMatrix[0]<< std::endl;
		
*/

  /*   dim3 thread2ds(TILE_DIM,TILE_DIM);
       dim3 grid2ds((Individual_Num+TILE_DIM-1)/TILE_DIM, (MarkerBlock_Size+TILE_DIM-1)/TILE_DIM);*/
    
                   /*dim3 grid2ds1((Individual_Num+TILE_DIM-1)/TILE_DIM, (Individual_Num+TILE_DIM-1)/TILE_DIM);*/  
                     /*dim3 thread1ds(Block_Size);
			       dim3 grid1ds((Individual_Num*Individual_Num)/Block_Size);*/ 
			       
			       /* dim3 thread2ds(TILE_DIM,TILE_DIM);
                dim3 grid2ds((Individual_Num+TILE_DIM-1)/TILE_DIM, (MarkerBlock_Size+TILE_DIM-1)/TILE_DIM);*/
                
                 /*  dim3 grid2ds1((Individual_Num+TILE_DIM-1)/TILE_DIM, (Individual_Num+TILE_DIM-1)/TILE_DIM);*/
                   /*   dim3 thread1ds(Block_Size);
		        dim3 grid1ds((Individual_Num*Individual_Num)/Block_Size);*/
		        /*  dim3 thread1ds(Block_Size);
	        dim3 grid1ds((Individual_Num*Individual_Num)/Block_Size);*/
       
  return;
}
